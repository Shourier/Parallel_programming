#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "lab1_func.h"

#include <chrono>
#include <iostream>
#include <vector>

#include <unordered_set>

using namespace std;
using namespace chrono;

const int group_number = 514;
const int my_number = 17;

const int X = group_number + my_number;
const int A = X % 4;						// A = 3
const int B = 7 + X % 7;					// B = 13

const int block_size = 512;					// Число нитей в блоке

#pragma region CUDA_processing 

__global__ void CUDA_processing(int* array, int* indices, int* borders)
{
	int start_index = 0;
	int end_index = borders[0];
	for (int i = 0; i < blockIdx.x; i++)
	{
		start_index += borders[i];
		end_index += borders[i];
	}

	__syncthreads();

	for (int i = start_index + threadIdx.x; i < end_index; i += block_size)
	{
		array[indices[i]] = (indices[i] * array[indices[i]]) & 255;
	}
}

#pragma endregion

int main()
{
	try
	{
		cout << "A = " << A << "; B = " << B << endl;

		int* array = open_jpg_file_as_binary_array();
		int* array_copy = new int[binary_array_size];
		for (int j = 0; j < binary_array_size; j++)
		{
			array_copy[j] = array[j];
		}


		vector<vector<int>> ind_matrix;
		for (int i = 0; i < B; i++)
		{
			vector<int> block_indeces;
			for (int j = binary_array_size - 1 - i; j >= 0; j -= B)
			{
				block_indeces.push_back(j);
			}
			ind_matrix.push_back(block_indeces);
		}


		int ind_arr_size = 0;
		int* indeces_borders = new int[B];
		for (int i = 0; i < B; i++)
		{
			ind_arr_size += ind_matrix[i].size();
			indeces_borders[i] = ind_matrix[i].size();
		}
		
		int* ind_arr = new int[ind_arr_size];
		int prev_sum = 0;
		for (int i = 0; i < B; i++)
		{
			for (int j = prev_sum; j < prev_sum + indeces_borders[i]; j++)
			{
				ind_arr[j] = ind_matrix[i][j - prev_sum];
			}
			prev_sum += indeces_borders[i];
		}


		int* gpu_data;
		int* gpu_indices;
		int* gpu_indices_borders;

		hipMalloc(&gpu_data, binary_array_size * sizeof(int));
		hipMalloc(&gpu_indices, ind_arr_size * sizeof(int));
		hipMalloc(&gpu_indices_borders, B * sizeof(int));

		hipMemcpy(gpu_data, array_copy, binary_array_size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(gpu_indices, ind_arr, ind_arr_size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(gpu_indices_borders, indeces_borders, B * sizeof(int), hipMemcpyHostToDevice);

		auto start = high_resolution_clock::now();

		CUDA_processing<<<B, block_size>>>(gpu_data, gpu_indices, gpu_indices_borders);
		hipDeviceSynchronize();

		auto finish = high_resolution_clock::now();
		duration<float> proc_time = finish - start;

		hipMemcpy(array_copy, gpu_data, binary_array_size * sizeof(int), hipMemcpyDeviceToHost);
		write_data_in_new_file(array_copy, "processing_by_CUDA.txt");
		cout << "CUDA method time: " << proc_time.count() << endl;

		hipFree(gpu_data);
		hipFree(gpu_indices);
		hipFree(gpu_indices_borders);
	}
	catch (const exception&)
	{

	}
}
